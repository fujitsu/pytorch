#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/quantized/fake_quant_affine.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <cmath>

/* Fake quantize a tensor
Args:
  output: output tensor.
  input : input tensor.
  sc:  scale to quantize the input tensor to
  zero_point: zero_point
  quant_min: minimum quantized value
  quant_max: maximum quantized value
Returns:
  Fake quantized tensor (float dtype).
*/
namespace at {
namespace native {
#if 1 //Added by Flab (Y. Tamiya)
static inline __host__ __device__
float fake_convert_fp(float x, int ebits, int mbits, int ebias)
{
  if (x == 0.0f || std::isinf(x) || std::isnan(x)) {
    return x;
  } else {
    // Round to Nearest Even Algorithm
    //printf("[DEBUG] ebits=%d,mbits=%d,ebias=%d,x=%f\n",ebits,mbits,ebias,x);
    const int FP32_EBITS = 8;
    const int FP32_MBITS = 23;
#   define BIAS(ebits)     ((1 << ((ebits) -1)) -1)
    uint32_t e_min = BIAS(FP32_EBITS) - BIAS(ebits) + ebias;
    uint32_t e_max = e_min + (1 << ebits) -1;
    //printf("[DEBUG] e_min/max = %d/%d\n", e_min, e_max);
    union {
      uint32_t i;
      float    f;
    } t;
    t.f = x;
    //printf("[DEBUG] x = 0x%08x\n", t.i);

    uint32_t s = (t.i & (1<< (FP32_EBITS+FP32_MBITS))) != 0;
    uint32_t e = (t.i >> FP32_MBITS) & ((1 << FP32_EBITS) -1);
    uint32_t m  = t.i & (((1 << mbits) - 1) << (FP32_MBITS - mbits)); 
    uint32_t r0 = t.i &  (1 << (FP32_MBITS - mbits -1));
    uint32_t r1 = t.i & ((1 << (FP32_MBITS - mbits -1)) -1);

    if (FP32_MBITS - mbits <= 0
	|| r0 == 0
	|| (FP32_MBITS - mbits >= 2 && r1 == 0
	    && (m & (1 << (FP32_MBITS - mbits))) == 0)) {
      ; //floor: nop
    } else {
      // ceil
      if (m == (((1 << mbits) -1) << (FP32_MBITS - mbits))) {
	m = 0;
	e += 1;
      } else {
	m += (1 << (FP32_MBITS - mbits));
      }
    }
    //printf("[DEBUG] s=%d, e=%d, m=%d (before clip)\n", s, e, m);
    if (e > e_max) { e = e_max; m = ((1 << mbits) -1) << (FP32_MBITS - mbits); }
    else if (e < e_min) { e = 0; m = 0; }

    t.i = (s << (FP32_EBITS+FP32_MBITS)) | (e << FP32_MBITS) | m; 
    //printf("[DEBUG] y = 0x%08x\n", t.i);
    return t.f;
  }
}
#endif //Added by Flab (Y. Tamiya)

void fake_quantize_tensor_kernel_cuda(
    Tensor& output,
    const Tensor& input,
    float scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max) {
  // scalar type of this function is guaranteed to be float
//Removed by Flab (Y. Tamiya)//  float inv_scale = 1.0f / scale;
  auto iter = TensorIterator();
  iter.dont_compute_common_dtype();
  iter.add_output(output);
  iter.add_input(input);
  iter.build();
#if 1 //Added by Flab (Y. Tamiya)
  if (scale == 0.0f) {
  gpu_kernel(iter,
    [zero_point] GPU_LAMBDA (float input_val) -> float {
       return fake_convert_fp(input_val, (zero_point>>8) & 0xff,
			      zero_point & 0xff,
			      (signed char)((zero_point>>16) & 0xff));
    });
  } else {
  float inv_scale = 1.0f / scale;
#endif //Added by Flab (Y. Tamiya)
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float input_val) -> float {
      return (fminf(
                quant_max,
                fmaxf(
                    quant_min,
                    static_cast<int64_t>(std::nearbyint(
                        input_val * inv_scale + zero_point)))) -
            zero_point) *
          scale;
    });
#if 1 //Added by Flab (Y. Tamiya)
  }
#endif //Added by Flab (Y. Tamiya)
}

void fake_quantize_grad_tensor_kernel_cuda(
    Tensor& input_grad,
    const Tensor& input,
    const Tensor& output_grad,
    float scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max) {
  // scalar type of this function is guaranteed to be float
//Removed by Flab (Y. Tamiya)//  float inv_scale = 1.0f / scale;
  auto iter = TensorIterator();
  iter.dont_compute_common_dtype();
  iter.add_output(input_grad);
  iter.add_input(output_grad);
  iter.add_input(input);
  iter.build();
#if 1 //Added by Flab (Y. Tamiya)
  if (scale == 0.0f) {
  gpu_kernel(iter,
    [zero_point] GPU_LAMBDA (float dy, float x) -> float {
       return fake_convert_fp(dy, (zero_point>>8) & 0xff,
			      zero_point & 0xff,
			      (signed char)((zero_point>>16) & 0xff));
    });
  } else {
  float inv_scale = 1.0f / scale;
#endif //Added by Flab (Y. Tamiya)
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float dy, float x) -> float {
      int64_t Xq = std::nearbyint(x * inv_scale + zero_point);
      return (Xq >= quant_min && Xq <= quant_max) * dy;
    });
#if 1 //Added by Flab (Y. Tamiya)
  }
#endif //Added by Flab (Y. Tamiya)
}

REGISTER_DISPATCH(fake_quant_tensor_stub, &fake_quantize_tensor_kernel_cuda);
REGISTER_DISPATCH(fake_quant_grad_tensor_stub, &fake_quantize_grad_tensor_kernel_cuda);

// Fake quantize per channel

void fake_quant_per_channel_cuda(TensorIterator &iter, int64_t quant_min, int64_t quant_max) {
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float input_val, float scale, int64_t zero_point) -> float {
#if 1 //Added by Flab (Y. Tamiya)
    if (scale == 0.0f) {
      return fake_convert_fp(input_val, (zero_point>>8) & 0xff,
			     zero_point & 0xff,
			     (signed char)((zero_point>>16) & 0xff));
    } else {
#endif //Added by Flab (Y. Tamiya)
      float inv_scale = 1.0f / scale;
      return (fminf(
                quant_max,
                fmaxf(
                    quant_min,
                    static_cast<int64_t>(std::nearbyint(
                        input_val * inv_scale + zero_point)))) -
            zero_point) *
          scale;
#if 1 //Added by Flab (Y. Tamiya)
    }
#endif //Added by Flab (Y. Tamiya)
    });
}

void fake_quant_grad_per_channel_cuda(TensorIterator &iter, int64_t quant_min, int64_t quant_max) {
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float x, float dy, float scale, int64_t zero_point) -> float {
#if 1 //Added by Flab (Y. Tamiya)
    if (scale == 0.0f) {
      return fake_convert_fp(dy, (zero_point>>8) & 0xff,
			     zero_point & 0xff,
			     (signed char)((zero_point>>16) & 0xff));
    } else {
#endif //Added by Flab (Y. Tamiya)
      float inv_scale = 1.0f / scale;
      int64_t Xq = std::nearbyint(x * inv_scale + zero_point);
      return (Xq >= quant_min && Xq <= quant_max) * dy;
#if 1 //Added by Flab (Y. Tamiya)
    }
#endif //Added by Flab (Y. Tamiya)
    });
}

REGISTER_DISPATCH(fake_quant_per_channel_stub, &fake_quant_per_channel_cuda);
REGISTER_DISPATCH(fake_quant_grad_per_channel_stub, &fake_quant_grad_per_channel_cuda);

} // namespace native
} // namespace at

#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/quantized/fake_quant_affine.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <cmath>

/* Fake quantize a tensor
Args:
  output: output tensor.
  input : input tensor.
  sc:  scale to quantize the input tensor to
  zero_point: zero_point
  quant_min: minimum quantized value
  quant_max: maximum quantized value
Returns:
  Fake quantized tensor (float dtype).
*/
namespace at {
namespace native {
#if 1 //Added by Flab (Y. Tamiya)
// Stochastic rounding from x to FlexFp(ebits, mbits, ebias) 
//   with random value of rnd (-0.5 <= rnd <= 0.5).
static inline __host__ __device__
float fake_convert_fp(float x, float rnd, int ebits, int mbits, int ebias)
{
  if (x == 0.0f || std::isinf(x) || std::isnan(x)) {
    return x;
  } else {
    // Round to Nearest Even Algorithm
    //printf("[DEBUG] ebits=%d,mbits=%d,ebias=%d,x=%f\n",ebits,mbits,ebias,x);
    const int FP32_EBITS = 8;
    const int FP32_MBITS = 23;
#   define BIAS(ebits)     ((1 << ((ebits) -1)) -1)
    uint32_t e_min = BIAS(FP32_EBITS) - BIAS(ebits) + ebias;
    uint32_t e_max = e_min + (1 << ebits) -1;
    //printf("[DEBUG] e_min/max = %d/%d\n", e_min, e_max);
    union {
      uint32_t i;
      float    f;
    } t, u;
    u.f = x;

    uint32_t us = (u.i & (1<< (FP32_EBITS+FP32_MBITS))) != 0;
    uint32_t ue = (u.i >> FP32_MBITS) & ((1 << FP32_EBITS) -1);
    if (ue < e_min) ue = e_min;
    u.i = (us << (FP32_EBITS+FP32_MBITS)) | (ue << FP32_MBITS);
    u.f /= float(1 << mbits); // unit val of the exponent.

    t.f = x + u.f * (rnd + 0.5); // stochastic rounding on mbits.
    //printf("[DEBUG] x = 0x%08x\n", t.i);

    uint32_t s = us;
    uint32_t e = (t.i >> FP32_MBITS) & ((1 << FP32_EBITS) -1);

    if (e < e_min) {
      // subnormal (denormalized) number
      mbits -= (e_min - e);
      if (mbits < 0) {
        //round to zero (e = 0, m = 0)
        t.i = (s << (FP32_EBITS+FP32_MBITS));
        return t.f;
      }
    }

    uint32_t m  = t.i & (((1 << mbits) - 1) << (FP32_MBITS - mbits));
    if (e > e_max) {
      // saturated toward Inf
      e = e_max; m = ((1 << mbits) -1) << (FP32_MBITS - mbits);
    }

    t.i = (s << (FP32_EBITS+FP32_MBITS)) | (e << FP32_MBITS) | m;
    //printf("[DEBUG] y = 0x%08x\n", t.i);
    return t.f;
  }
}
#endif //Added by Flab (Y. Tamiya)

void fake_quantize_tensor_kernel_cuda(
    Tensor& output,
    const Tensor& input,
    float scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max,
    bool train) {
  // scalar type of this function is guaranteed to be float
  //Moved by Flab (Y. Tamiya)//float inv_scale = 1.0f / scale;
  // uniform(-.5, .5) random values for stochastic rounding. (Added by Flab)
  Tensor rnd = train ? input.new_empty(input.sizes()).uniform_(-.5, .5).detach_() :
                       input.new_full(input.sizes(), 0.).detach_();
  auto iter = TensorIteratorConfig()
    .check_all_same_dtype(false)
    .add_output(output)
    .add_input(input)
    .add_input(rnd)
    .build();
#if 1 //Added by Flab (Y. Tamiya)
  if (std::isnan(scale)) {
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float input_val, float rnd) -> float {
       return fake_convert_fp(input_val, rnd, (zero_point>>8) & 0xff,
			      zero_point & 0xff,
			      (signed char)((zero_point>>16) & 0xff));
    });
  } else {
  float inv_scale = 1.0f / scale;
#endif //Added by Flab (Y. Tamiya)
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float input_val, float rnd) -> float {
      return (fminf(
                quant_max,
                fmaxf(
                    quant_min,
                    // use stochastic rounding (by Flab)
                    static_cast<int64_t>(std::nearbyint(
                        input_val * inv_scale + zero_point + rnd)))) -
            zero_point) *
          scale;
    });
#if 1 //Added by Flab (Y. Tamiya)
  }
#endif //Added by Flab (Y. Tamiya)
}

void fake_quantize_grad_tensor_kernel_cuda(
    Tensor& input_grad,
    const Tensor& input,
    const Tensor& output_grad,
    float scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max) {
  // scalar type of this function is guaranteed to be float
  float inv_scale = 1.0f / scale;
  auto iter = TensorIteratorConfig()
    .check_all_same_dtype(false)
    .add_output(input_grad)
    .add_input(output_grad)
    .add_input(input)
    .build();
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float dy, float x) -> float {
      int64_t Xq = std::nearbyint(x * inv_scale + zero_point);
      return (Xq >= quant_min && Xq <= quant_max) * dy;
    });
}

REGISTER_DISPATCH(fake_quant_tensor_stub, &fake_quantize_tensor_kernel_cuda);
REGISTER_DISPATCH(fake_quant_grad_tensor_stub, &fake_quantize_grad_tensor_kernel_cuda);

// Fake quantize per channel

void fake_quant_per_channel_cuda(TensorIterator &iter, int64_t quant_min, int64_t quant_max) {
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float input_val, float rnd, float scale, int64_t zero_point) -> float {
#if 1 //Added by Flab (Y. Tamiya)
    if (std::isnan(scale)) {
      return fake_convert_fp(input_val, rnd, (zero_point>>8) & 0xff,
			     zero_point & 0xff,
			     (signed char)((zero_point>>16) & 0xff));
    } else {
#endif //Added by Flab (Y. Tamiya)
      float inv_scale = 1.0f / scale;
      return (fminf(
                quant_max,
                fmaxf(
                    quant_min,
                    // use stochastic rounding (by Flab)
                    static_cast<int64_t>(std::nearbyint(
                        input_val * inv_scale + zero_point + rnd)))) -
            zero_point) *
          scale;
#if 1 //Added by Flab (Y. Tamiya)
    }
#endif //Added by Flab (Y. Tamiya)
    });
}

void fake_quant_grad_per_channel_cuda(TensorIterator &iter, int64_t quant_min, int64_t quant_max) {
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float x, float dy, float scale, int64_t zero_point) -> float {
      float inv_scale = 1.0f / scale;
      int64_t Xq = std::nearbyint(x * inv_scale + zero_point);
      return (Xq >= quant_min && Xq <= quant_max) * dy;
    });
}

REGISTER_DISPATCH(fake_quant_per_channel_stub, &fake_quant_per_channel_cuda);
REGISTER_DISPATCH(fake_quant_grad_per_channel_stub, &fake_quant_grad_per_channel_cuda);

} // namespace native
} // namespace at

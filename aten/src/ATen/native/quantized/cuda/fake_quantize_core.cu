#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/quantized/fake_quant_affine.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <thrust/tuple.h>
#include <cmath>

/* Fake quantize a tensor
Args:
  output: output tensor.
  input : input tensor.
  sc:  scale to quantize the input tensor to
  zero_point: zero_point
  quant_min: minimum quantized value
  quant_max: maximum quantized value
Returns:
  Fake quantized tensor (float dtype).
*/
namespace at {
namespace native {

// Stochastic rounding from x to FlexFp(ebits, mbits, ebias) 
//   with random value of rnd (-0.5 <= rnd <= 0.5).
static inline __host__ __device__
float fake_convert_fp(float x, float rnd, int ebits, int mbits, int ebias)
{
  if (x == 0.0f || std::isinf(x) || std::isnan(x)) {
    return x;
  } else {
    // Round to Nearest Even Algorithm
    //printf("[DEBUG] ebits=%d,mbits=%d,ebias=%d,x=%f\n",ebits,mbits,ebias,x);
    const int FP32_EBITS = 8;
    const int FP32_MBITS = 23;
#   define BIAS(ebits)     ((1 << ((ebits) -1)) -1)
    uint32_t e_min = BIAS(FP32_EBITS) - BIAS(ebits) + ebias;
    uint32_t e_max = e_min + (1 << ebits) -1;
    //printf("[DEBUG] e_min/max = %d/%d\n", e_min, e_max);
    union {
      uint32_t i;
      float    f;
    } t, u;
    u.f = x;

    uint32_t us = (u.i & (1<< (FP32_EBITS+FP32_MBITS))) != 0;
    uint32_t ue = (u.i >> FP32_MBITS) & ((1 << FP32_EBITS) -1);
    if (ue < e_min) ue = e_min;
    u.i = (us << (FP32_EBITS+FP32_MBITS)) | (ue << FP32_MBITS);
    u.f /= float(1 << mbits); // unit val of the exponent.

    t.f = x + u.f * (rnd + 0.5); // stochastic rounding on mbits.
    //printf("[DEBUG] x = 0x%08x\n", t.i);

    uint32_t s = us;
    uint32_t e = (t.i >> FP32_MBITS) & ((1 << FP32_EBITS) -1);

    if (e < e_min) {
      // subnormal (denormalized) number
      mbits -= (e_min - e);
      if (mbits < 0) {
        //round to zero (e = 0, m = 0)
        t.i = (s << (FP32_EBITS+FP32_MBITS));
        return t.f;
      }
    }

    uint32_t m  = t.i & (((1 << mbits) - 1) << (FP32_MBITS - mbits));
    if (e > e_max) {
      // saturated toward Inf
      e = e_max; m = ((1 << mbits) -1) << (FP32_MBITS - mbits);
    }

    t.i = (s << (FP32_EBITS+FP32_MBITS)) | (e << FP32_MBITS) | m;
    //printf("[DEBUG] y = 0x%08x\n", t.i);
    return t.f;
  }
}

void fake_quantize_tensor_kernel_cuda(
    Tensor& output,
    const Tensor& input,
    float scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max,
    bool train) {
  // scalar type of this function is guaranteed to be float
  //Moved by Fujitsu//float inv_scale = 1.0f / scale;
  // uniform(-.5, .5) random values for stochastic rounding. (Added by Fujitsu)
  Tensor rnd = train ? input.new_empty(input.sizes()).uniform_(-.5, .5).detach_() :
                       input.new_full(input.sizes(), 0.).detach_();
  auto iter = TensorIteratorConfig()
    .check_all_same_dtype(false)
    .add_output(output)
    .add_input(input)
    .add_input(rnd)
    .build();
  if (std::isnan(scale)) {
    gpu_kernel(iter,
      [=] GPU_LAMBDA (float input_val, float rnd) -> float {
        return fake_convert_fp(input_val, rnd, (zero_point>>8) & 0xff,
			      zero_point & 0xff,
			      (signed char)((zero_point>>16) & 0xff));
    });
  } else {
    float inv_scale = 1.0f / scale;
    gpu_kernel(iter,
      [=] GPU_LAMBDA (float input_val, float rnd) -> float {
        return (fminf(
                quant_max,
                fmaxf(
                    quant_min,
                    // use stochastic rounding (by Fujitsu)
                    static_cast<int64_t>(std::nearbyint(
                        input_val * inv_scale + zero_point + rnd)))) -
            zero_point) *
          scale;
    });
  }
}

void fake_quantize_grad_tensor_kernel_cuda(
    Tensor& input_grad,
    const Tensor& input,
    const Tensor& output_grad,
    float scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max) {
  // scalar type of this function is guaranteed to be float
  float inv_scale = 1.0f / scale;
  auto iter = TensorIteratorConfig()
    .check_all_same_dtype(false)
    .add_output(input_grad)
    .add_input(output_grad)
    .add_input(input)
    .build();
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float dy, float x) -> float {
      int64_t Xq = std::nearbyint(x * inv_scale + zero_point);
      return (Xq >= quant_min && Xq <= quant_max) * dy;
    });
}

void _fake_quantize_grad_learnable_tensor_kernel_cuda(
    TensorIterator& iter,
    float scale,
    float inv_scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max) {
  float dscale_small = quant_min - zero_point;
  float dscale_big = quant_max - zero_point;
  gpu_kernel_multiple_outputs(
    iter, [=] GPU_LAMBDA (float XInput, float dYInput) -> thrust::tuple<float, float, float> {
      float dXOutput, dZeroPointOutput, dScaleOutput;
      int64_t xq = std::nearbyint(zero_point + XInput * inv_scale);
      dXOutput = dYInput * (xq >= quant_min && xq <= quant_max);
      xq = std::max(std::min(xq, quant_max), quant_min);
      float xfq = static_cast<float>((xq - zero_point) * scale);
      if (xq == quant_min || xq == quant_max) {
        dZeroPointOutput = (dYInput) * (-1) * scale;
        dScaleOutput = (xq == quant_min) ? (dYInput * dscale_small) : (dYInput * dscale_big);
      } else {
        dZeroPointOutput = 0;
        dScaleOutput = (dYInput) * (xfq - (XInput)) * inv_scale;
      }
      return {dXOutput, dScaleOutput, dZeroPointOutput};
  });
}

REGISTER_DISPATCH(fake_quant_tensor_stub, &fake_quantize_tensor_kernel_cuda);
REGISTER_DISPATCH(fake_quant_grad_tensor_stub, &fake_quantize_grad_tensor_kernel_cuda);
REGISTER_DISPATCH(fake_quant_grad_learnable_tensor_stub, &_fake_quantize_grad_learnable_tensor_kernel_cuda);

// Fake quantize per channel

void fake_quant_per_channel_cuda(TensorIterator &iter, int64_t quant_min, int64_t quant_max) {
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float input_val, float rnd, float scale, int64_t zero_point) -> float {
      if (std::isnan(scale)) {
        return fake_convert_fp(input_val, rnd, (zero_point>>8) & 0xff,
			     zero_point & 0xff,
			     (signed char)((zero_point>>16) & 0xff));
      } else {
        float inv_scale = 1.0f / scale;
        return (fminf(
                quant_max,
                fmaxf(
                    quant_min,
                    // use stochastic rounding (by Flab)
                    static_cast<int64_t>(std::nearbyint(
                        input_val * inv_scale + zero_point + rnd)))) -
            zero_point) *
          scale;
      }
    });
}

void fake_quant_grad_per_channel_cuda(TensorIterator &iter, int64_t quant_min, int64_t quant_max) {
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float x, float dy, float scale, int64_t zero_point) -> float {
      float inv_scale = 1.0f / scale;
      int64_t Xq = std::nearbyint(x * inv_scale + zero_point);
      return (Xq >= quant_min && Xq <= quant_max) * dy;
    });
}

void _fake_quantize_grad_learnable_channel_kernel_cuda(TensorIterator &iter, int64_t quant_min, int64_t quant_max) {
  gpu_kernel_multiple_outputs(iter,
    [=] GPU_LAMBDA (float x_input, float dy_input, float scale_input, float zero_point_input) -> thrust::tuple<float, float, float> {
      float dx_output, dscale_output, dzero_point_output;
      float inv_scale = 1.0f / scale_input;
      float dscale_small = quant_min - zero_point_input;
      float dscale_big = quant_max - zero_point_input;
      // Calculate gradients for X.
      int64_t xqi = std::nearbyint(zero_point_input + x_input * inv_scale);
      dx_output = dy_input * (xqi >= quant_min && xqi <= quant_max);
      // Calculate gradients for scale and zero point.
      xqi = std::max(std::min(xqi, quant_max), quant_min);
      float xfqi = static_cast<float>((xqi - zero_point_input) * scale_input);
      if (xqi == quant_min || xqi == quant_max) {
        dzero_point_output = dy_input * (-1) * scale_input;
        dscale_output = (xqi == quant_min) ? (dy_input * dscale_small) : (dy_input * dscale_big);
      } else {
        dzero_point_output = 0;
        dscale_output = dy_input * (xfqi - x_input) * inv_scale;
      }
      return {dx_output, dscale_output, dzero_point_output};
    });
}

REGISTER_DISPATCH(fake_quant_per_channel_stub, &fake_quant_per_channel_cuda);
REGISTER_DISPATCH(fake_quant_grad_per_channel_stub, &fake_quant_grad_per_channel_cuda);
REGISTER_DISPATCH(fake_quant_grad_learnable_channel_stub, &_fake_quantize_grad_learnable_channel_kernel_cuda);

} // namespace native
} // namespace at
